#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"


// return GB/sec
float GBPerSec(int bytes, float sec) {
  return static_cast<float>(bytes) / (1024. * 1024. * 1024.) / sec;
}


// This is the CUDA "kernel" function that is run on the GPU.  You
// know this because it is marked as a __global__ function.
__global__ void
saxpy_kernel(int N, float alpha, float* x, float* y, float* result) {

    // compute overall thread index from position of thread in current
    // block, and given the block we are in (in this example only a 1D
    // calculation is needed so the code only looks at the .x terms of
    // blockDim and threadIdx.
    int index = blockIdx.x * blockDim.x + threadIdx.x;


    // this check is necessary to make the code work for values of N
    // that are not a multiple of the thread block size (blockDim.x)
    if (index < N)
       result[index] = alpha * x[index] + y[index];
}


// saxpyCuda --
//
// This function is regular C code running on the CPU.  It allocates
// memory on the GPU using CUDA API functions, uses CUDA API functions
// to transfer data from the CPU's memory address space to GPU memory
// address space, and launches the CUDA kernel function on the GPU.
void saxpyCuda(int N, float alpha, float* xarray, float* yarray, float* resultarray) {

    // must read both input arrays (xarray and yarray) and write to
    // output array (resultarray)
    int totalBytes = sizeof(float) * 3 * N;

    // compute number of blocks and threads per block.  In this
    // application we've hardcoded thread blocks to contain 512 CUDA
    // threads.
    const int threadsPerBlock = 512;

    // Notice the round up here.  The code needs to compute the number
    // of threads blocks needed such that there is one thread per
    // element of the arrays.  This code is written to work for values
    // of N that are not multiples of threadPerBlock.
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    // These are pointers that will be pointers to memory allocated
    // *one the GPU*.  You should allocate these pointers via
    // hipMalloc.  You can access the resulting buffers from CUDA
    // device kernel code (see the kernel function saxpy_kernel()
    // above) but you cannot access the contents these buffers from
    // this thread. CPU threads cannot issue loads and stores from GPU
    // memory!
    float* device_x = nullptr;
    float* device_y = nullptr;
    float* device_result = nullptr;

    float *pinned_x = nullptr;
    float *pinned_y = nullptr;
    float *pinned_res = nullptr;

    hipHostAlloc(&pinned_x, sizeof(float) * N, hipHostMallocDefault);
    hipHostAlloc(&pinned_y, sizeof(float) * N, hipHostMallocDefault);
    hipHostAlloc(&pinned_res, sizeof(float) * N, hipHostMallocDefault);

    memcpy(pinned_x, xarray, sizeof(float) * N);
    memcpy(pinned_y, yarray, sizeof(float) * N);

    hipMalloc(&device_x, sizeof(float) * N);
    hipMalloc(&device_y, sizeof(float) * N);
    hipMalloc(&device_result, sizeof(float) * N);
        
    // start timing after allocation of device memory
    double startTime = CycleTimer::currentSeconds();

    hipMemcpy(device_x, pinned_x, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(device_y, pinned_y, sizeof(float) * N, hipMemcpyHostToDevice);

    double startRun = CycleTimer::currentSeconds();
   
    // run CUDA kernel. (notice the <<< >>> brackets indicating a CUDA
    // kernel launch) Execution on the GPU occurs here.
    saxpy_kernel<<<blocks, threadsPerBlock>>>(N, alpha, device_x, device_y, device_result);
    hipDeviceSynchronize();

    double endRun = CycleTimer::currentSeconds();

    hipMemcpy(resultarray, device_result, sizeof(float) * N, hipMemcpyDeviceToHost);

    double endTime = CycleTimer::currentSeconds();
    
    // end timing after result has been copied back into host memory

    memcpy(resultarray, pinned_res, sizeof(float) * N);

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n",
		errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    double runDuration = endRun - startRun;
    printf("Effective BW by CUDA saxpy: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, GBPerSec(totalBytes, overallDuration - runDuration));
    printf("Run time: %.3f\n", 1000.f * runDuration);

    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_result);
    
}

void printCudaInfo() {

    // print out stats about the GPU in the machine.  Useful if
    // students want to know what GPU they are running on.

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
        printf("   Shared mem per block: %.0f KB\n", static_cast<float>(deviceProps.sharedMemPerBlock));
    }
    printf("---------------------------------------------------------\n");
}
